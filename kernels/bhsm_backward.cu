
#include <hip/hip_runtime.h>
extern "C"

__global__ void bhsm_backward(
  const float *wxy,
  const float *x,
  const float *w,
  const int   *ts,
  const int   *paths,
  const float *codes,
  const int   *begins,
  const float *gLoss,
  const int   n_in,
  const int   max_len,
  const int   n_ex,
  float *gx,
  float *gW
) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n_ex * max_len) {
    int idx = i / max_len;
    int offset = i - idx * max_len;
    int t = ts[idx];

    int begin = begins[t];
    int length = begins[t+1] - begin;

    if (offset < length) {
      int p = begin + offset;
      int node = paths[p];
      float g = -gLoss[0] * codes[p] / (1.0f + exp(wxy[i]));

      int w_start = n_in * node;
      int x_start = n_in * idx;
      for (int j = 0; j < n_in; ++j) {
        int w_i = w_start + j;
        int x_i = x_start + j;
        atomicAdd(gx + x_i, g * w[w_i]);
        atomicAdd(gW + w_i, g * x[x_i]);
      }
    }
  }
}
