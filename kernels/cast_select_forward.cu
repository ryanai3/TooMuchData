#include "hip/hip_runtime.h"
extern "C"

__global__ void cast_select_forward(
  const float * actors,
  const int   * begins,
  const int   * offsets,
  const float * out
) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  int offset = offsets[i];
  if (offset != -1) {
    int variable_idx = begins[i] + offset[i];

  }
  

}
