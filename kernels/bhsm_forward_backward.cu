
#include <hip/hip_runtime.h>
extern "C"

__global__ void bhsm_forward_backward(
  const float *x,
  const float *w,
  const int   *ts,
  const int   *paths,
  const float *codes,
  const int   *begins,
  const int   n_in,
  const int   max_len,
  const int   n_ex,
  float *ls,
  float *gx,
  float *gW
) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n_ex * max_len) {
    int idx = i / max_len;
    int offset = i - idx * max_len;
    int t = ts[idx];

    int begin = begins[t];
    int length = begins[t+1] - begin;

    if (offset < length) {
      int p = begin + offset;
      int node = paths[p];

      float wx = 0;
      int w_start = n_in * node;
      int x_start = n_in * idx;
      for (int j = 0; j < n_in; ++j) {
//        int w_i = w_start + j;
//        int x_i = x_start + j;
//        wx += (w[w_i] * x[x_i]);
        wx +=(w[w_start + j] * x[x_start + j]);
      }
      wx *= codes[p];
      float g = -codes[p] / (1.0f + exp(wx)); 
      ls[i] = log(1 + exp(-wx));
      for (int j = 0; j < n_in; ++j) {
        int w_i = w_start + j;
        int x_i = x_start + j;
//        gx[x_i] += g * w[w_i];
//        gW[w_i] += g * x[x_i];
        atomicAdd(gx + x_i, g * w[w_i]);
        atomicAdd(gW + w_i, g * x[x_i]);

      }
    }
  }
}


