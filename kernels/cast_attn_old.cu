#include "hip/hip_runtime.h"
extern "C"

__global__ void attend_forward(
  const float *actors,
  const int   *begins,
  const int   batch_size,
  const int   *cast_sizes,
  const int   *vec_size,
  const float *query,
  float       *scores
) {

  int batch_i = blockIdx.x * blockDim.x + threadIdx.x;
  int cast_i  = blockIdx.y * blockDim.y + threadIdx.y;

  if (batch_i < n_actors) {
    if (cast_i < cast_sizes[batch_i]) {
      for (int i = 0; i < vec_size; ++i) {
        int idx = begins[batch_i] + cast_i;
        scores[idx] += actors[(idx * vec_size) + i] * query[batch_i + i];
      }   
    }
  }
}
